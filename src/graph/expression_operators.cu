#include "hip/hip_runtime.h"
// This file is part of the Marian toolkit.

//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "graph/expression_operators.h"
#include "graph/node_operators.h"

namespace marian {

Expr debug(Expr a, const std::string& message) {
  a->debug(message);
  return a;
}

Expr name(Expr a, const std::string& name) {
  a->set_name(name);
  a->graph()->add_named_node(a, name);
  return a;
}

Expr rows(Expr a, const std::vector<size_t>& indeces) {
  return Expression<RowsNodeOp>(a, indeces);
}

Expr logit(Expr a) {
  return Expression<LogitNodeOp>(a);
}

Expr relu(Expr a) {
  return Expression<ReLUNodeOp>(a);
}

Expr log(Expr a) {
  return Expression<LogNodeOp>(a);
};

Expr exp(Expr a) {
  return Expression<ExpNodeOp>(a);
};

Expr operator-(Expr a) {
  return Expression<NegNodeOp>(a);
};

Expr softmax(Expr a, Expr mask) {
  return Expression<SoftmaxNodeOp>(a, mask);
}

Expr logsoftmax(Expr a) {
  return Expression<LogSoftmaxNodeOp>(a);
}

/*********************************************************/

Expr operator+(Expr a, Expr b) {
  return Expression<PlusNodeOp>(a, b);
}

Expr operator-(Expr a, Expr b) {
  return Expression<MinusNodeOp>(a, b);
}

Expr operator*(Expr a, Expr b) {
  return Expression<MultNodeOp>(a, b);
}

Expr operator/(Expr a, Expr b) {
  return Expression<DivNodeOp>(a, b);
}

Expr dot(Expr a, Expr b) {
  return Expression<DotNodeOp>(a, b);
}

Expr transpose(Expr a) {
  return Expression<TransposeNodeOp>(a);
}

Expr step(Expr a, size_t step) {
  return Expression<TimestepNodeOp>(a, step);
}

Expr cross_entropy(Expr a, Expr b) {
  auto sOrig = a->shape();
  auto sOut = a->shape();
  Shape sTemp({sOrig[0] * sOrig[2] * sOrig[3], sOrig[1], 1, 1});
  sOut.set(1, 1);
  return reshape(Expression<CrossEntropyNodeOp>(reshape(a, sTemp), b), sOut);
}

Expr affine(Expr a, Expr b, Expr c) {
  std::vector<Expr> nodes = {a, b, c};
  return Expression<AffineNodeOp>(nodes);
}

Expr plus(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr tanh(const std::vector<Expr>& nodes) {
  return Expression<TanhNodeOp>(nodes);
}

Expr logit(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr relu(const std::vector<Expr>&) {
  UTIL_THROW2("Not implemented");
}

Expr sqrt(Expr a, float eps) {
  return Expression<SqrtNodeOp>(a, eps);
}

Expr square(Expr a) {
  return Expression<SquareNodeOp>(a);
}

Expr layer_norm(Expr x, Expr gamma, Expr beta) {
  std::vector<Expr> nodes = {x, gamma};
  if(beta)
    nodes.push_back(beta);
  return Expression<LayerNormalizationOp>(nodes);
}

Expr convolution(Expr x, Expr filters, int hPad, int wPad) {
  std::vector<Expr> nodes = {x, filters};
  return Expression<ConvolutionOp>(nodes, hPad, wPad);
}

Expr max_pooling(Expr x, int hPad, int wPad) {
  return Expression<MaxPoolingOp>(x, hPad, wPad);
}

//Expr batch_norm(Expr x, Expr gamma, Expr beta) {
//  auto mju = mean(x, keywords::axis=0);
//  auto xmmju = x - mju;
//  auto std = sqrt(mean(square(xmmju), keywords::axis=0), 1e-9);
//
//  if(beta)
//    return gamma * (xmmju / std) + beta;
//  else
//    return gamma * (xmmju / std);
//}

}
