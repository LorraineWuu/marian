#include "graph/expression_graph.h"
#include "graph/node.h"
#include "kernels/tensor_operators.h"

namespace marian {

size_t Node::allocate(bool fake) {
  size_t elements = 0;
  if(!val_) {
    graph_->tensor(val_, shape_, fake);
    elements = val_->shape().elements();
  }
  return elements;
}

void Node::free(bool fake) {
  if(val_)
    graph_->free(val_, fake);
  if(adj_)
    graph_->free(adj_, fake);
}

void Node::init_dependent(bool fake) {
  if(!adj_) {
    graph_->tensor(adj_, shape_, fake);
    if(!fake)
      adj_->set(1);
  }
}

void Node::set_zero_adjoint(bool fake) {
  if(!adj_) {
    graph_->tensor(adj_, shape_, fake);
    if(!fake)
      adj_->set(0);
  }
}

float Node::scalar() {
  return val_->scalar();
}


hipblasHandle_t Node::getCublasHandle() {
  return graph_->getCublasHandle();
}

void NaryNodeOp::remove_children_from_top_nodes() {
  for(auto child : children_)
    graph_->remove_top_node(child);
}

}
