#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "marian.h"
#include "training/config.h"
#include "optimizers/optimizers.h"
#include "optimizers/clippers.h"
#include "data/batch_generator.h"
#include "data/corpus.h"
#include "models/multi_gnmt.h"
#include "translator/nth_element.h"
#include "common/history.h"


namespace marian {

template <class Builder>
class BeamSearch {
  private:
    Ptr<Builder> builder_;
    size_t beamSize_;
    hipStream_t stream_{0};

  public:
    BeamSearch(Ptr<Builder> builder)
     : builder_(builder),
       beamSize_(12)
    {}

    Beam toHyps(const std::vector<uint> keys,
                const std::vector<float> costs,
                size_t vocabSize,
                const Beam& beam) {
      Beam newBeam;
      for(int i = 0; i < keys.size(); ++i) {
        int embIdx = keys[i] % vocabSize;
        int hypIdx = keys[i] / vocabSize;
        float cost = costs[i];

        newBeam.push_back(
          New<Hypothesis>(beam[hypIdx], embIdx, hypIdx, cost));
      }
      return newBeam;
    }

    Beam pruneBeam(const Beam& beam) {
      Beam newBeam;
      for(auto hyp : beam) {
        if(hyp->GetWord() > 0) {
          newBeam.push_back(hyp);
        }
      }
      return newBeam;
    }

    std::tuple<std::vector<Expr>, Expr>
    step(std::vector<Expr> hyps,
         Ptr<EncoderState> encState,
         const std::vector<size_t> hypIdx = {},
         const std::vector<size_t> embIdx = {}) {
      using namespace keywords;
      auto graph = hyps[0]->graph();

      // @TODO: not hard-coded!
      int dimTrgEmb_ = 512;
      int dimTrgVoc_ = 50000;

      std::vector<Expr> selectedHyps;
      Expr selectedEmbs;
      if(embIdx.empty()) {
        selectedHyps = hyps;
        selectedEmbs = graph->constant(shape={1, dimTrgEmb_},
                                       init=inits::zeros);
      }
      else {
        // @TODO : solve this better than reshaping!
        for(auto h : hyps)
          selectedHyps.push_back(
            reshape(rows(h, hypIdx), {1, h->shape()[1], 1, (int)hypIdx.size()}));

        auto yEmb = Embedding("Wemb_dec", dimTrgVoc_, dimTrgEmb_)(graph);
        selectedEmbs = reshape(rows(yEmb, embIdx),
                               {1, yEmb->shape()[1], 1, (int)embIdx.size()});
      }

      Expr logits;
      std::vector<Expr> newHyps;
      std::tie(logits, newHyps) = builder_->step(selectedEmbs,
                                                 selectedHyps,
                                                 encState,
                                                 true);
      return std::make_tuple(newHyps, logsoftmax(logits));
    }

    std::tuple<std::vector<Expr>, Expr>
    step(std::vector<Expr> hyps,
         Ptr<EncoderState> encState,
         const Beam& beam) {

      std::vector<size_t> hypIndeces;
      std::vector<size_t> embIndeces;
      std::vector<float> beamCosts;

      for(auto hyp : beam) {
        hypIndeces.push_back(hyp->GetPrevStateIndex());
        embIndeces.push_back(hyp->GetWord());
        beamCosts.push_back(hyp->GetCost());
      }

      auto graph = hyps[0]->graph();
      auto costs = graph->constant(keywords::shape={1, 1, 1, (int)beamCosts.size()},
                                   keywords::init=inits::from_vector(beamCosts));

      std::vector<Expr> newHyps;
      Expr probs;
      std::tie(newHyps, probs) = step(hyps,
                                      encState,
                                      hypIndeces,
                                      embIndeces);
      probs = probs + costs;
      return std::make_tuple(newHyps, probs);
    }

    Ptr<History> search(Ptr<ExpressionGraph> graph,
                        Ptr<data::CorpusBatch> batch) {

      std::vector<Expr> startStates;
      Ptr<EncoderState> encState;
      std::tie(startStates, encState)
        = builder_->buildEncoder(graph, batch);

      size_t pos = 0;
      auto history = New<History>(0);
      Beam beam(1, New<Hypothesis>());
      bool first = true;
      bool final = false;
      std::vector<size_t> beamSizes(1, beamSize_);
      auto nth = New<NthElement>(beamSize_, batch->size(), stream_);

      history->Add(beam);

      std::vector<Expr> hyps;
      Expr probs;
      do {

        if(first) {
          std::tie(hyps, probs) = step(startStates, encState);
          pos = graph->forward();
        }
        else {
          std::tie(hyps, probs) = step(hyps, encState, beam);
          beamSizes[0] = beam.size();
          pos = graph->forward(pos);
        }

        size_t dimTrgVoc = probs->shape()[1];

        std::vector<unsigned> outKeys;
        std::vector<float> outCosts;

        for(int i = 0; i < probs->shape()[3]; i++) {
          probs->val()->set(i * dimTrgVoc + 1, std::numeric_limits<float>::lowest());
        }

        nth->getNBestList(beamSizes, probs->val(),
                          outCosts, outKeys, first);
        first = false;

        beam = toHyps(outKeys, outCosts, dimTrgVoc, beam);
        final = history->size() >= 3 * batch->words();
        history->Add(beam, final);
        beam = pruneBeam(beam);

      } while(!beam.empty() && !final);

      return history;
    }
};

}

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  auto options = New<Config>(argc, argv, false);

  std::vector<std::string> files =
    {"/work/wmt16/work/unbabel/marian2016/dev.mt",
    "/work/wmt16/work/unbabel/marian2016/dev.src"};
    //{"../benchmark/marian32K/test.txt"};

  std::vector<std::string> vocab =
    {"/work/wmt16/work/unbabel/marian2016/vocab.mt.json",
    "/work/wmt16/work/unbabel/marian2016/vocab.src.json"};

  YAML::Node& c = options->get();
  c["train-sets"] = files;
  c["vocabs"] = vocab;

  auto corpus = DataSet<Corpus>(options);
  BatchGenerator<Corpus> bg(corpus, options);

  auto graph = New<ExpressionGraph>();
  graph->setDevice(1);

  auto target = New<Vocab>();
  target->load("/work/wmt16/work/unbabel/marian2016/vocab.pe.json", 40000);

  auto encdec = New<MultiGNMT>(options);
  encdec->load(graph, "/work/wmt16/work/unbabel/marian2016/multisource/model.10000.npz");

  graph->reserveWorkspaceMB(128);

  boost::timer::cpu_timer timer;
  bg.prepare(false);
  while(bg) {
    auto batch = bg.next();
    auto search = New<BeamSearch<MultiGNMT>>(encdec);
    auto history = search->search(graph, batch);

    auto results = history->NBest(1);
    for(auto r : results) {
        for(auto w : r.first)
        if(w != 0)
          std::cout << (*target)[w] << " ";
      //std::cout << r.second->GetCost() << std::endl;
      std::cout << std::endl;
    }
  }
  std::cerr << timer.format(5, "%ws") << std::endl;

  return 0;

}
