#include <algorithm>
#include <chrono>
#include <iomanip>
#include <string>
#include <cstdio>
#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>

#include "marian.h"
#include "training/config.h"
#include "optimizers/optimizers.h"
#include "optimizers/clippers.h"
#include "data/batch_generator.h"
#include "data/corpus.h"
#include "models/gnmt.h"
#include "translator/nth_element.h"
#include "common/history.h"


namespace marian {

template <class Builder>
class BeamSearch {
  private:
    Ptr<Builder> builder_;
    size_t beamSize_;
    hipStream_t stream_{0};

  public:
    BeamSearch(Ptr<Builder> builder)
     : builder_(builder),
       beamSize_(12)
    {}

    Beam toHyps(const std::vector<uint> keys,
                const std::vector<float> costs,
                size_t vocabSize,
                const Beam& beam) {
      Beam newBeam;
      for(int i = 0; i < keys.size(); ++i) {
        int embIdx = keys[i] % vocabSize;
        int hypIdx = keys[i] / vocabSize;
        float cost = costs[i];

        newBeam.push_back(
          New<Hypothesis>(beam[hypIdx], embIdx, hypIdx, cost));
      }
      return newBeam;
    }

    Beam pruneBeam(const Beam& beam) {
      Beam newBeam;
      for(auto hyp : beam) {
        if(hyp->GetWord() > 0) {
          newBeam.push_back(hyp);
        }
      }
      return newBeam;
    }

    std::tuple<std::vector<Expr>, Expr>
    step(std::vector<Expr> hyps,
         Expr srcContext,
         Expr srcMask,
         const std::vector<size_t> hypIdx = {},
         const std::vector<size_t> embIdx = {}) {
      using namespace keywords;
      auto graph = hyps[0]->graph();

      // @TODO: not hard-coded!
      int dimTrgEmb_ = 512;
      int dimTrgVoc_ = 50000;

      std::vector<Expr> selectedHyps;
      Expr selectedEmbs;
      if(embIdx.empty()) {
        selectedHyps = hyps;
        selectedEmbs = graph->constant(shape={1, dimTrgEmb_},
                                       init=inits::zeros);
      }
      else {
        // @TODO : solve this better than reshaping!
        for(auto h : hyps)
          selectedHyps.push_back(
            reshape(rows(h, hypIdx), {1, h->shape()[1], 1, (int)hypIdx.size()}));

        auto yEmb = Embedding("Wemb_dec", dimTrgVoc_, dimTrgEmb_)(graph);
        selectedEmbs = reshape(rows(yEmb, embIdx),
                               {1, yEmb->shape()[1], 1, (int)embIdx.size()});
      }

      Expr logits;
      std::vector<Expr> newHyps;
      std::tie(logits, newHyps) = builder_->step(selectedEmbs,
                                                 selectedHyps,
                                                 srcContext,
                                                 srcMask,
                                                 true);
      return std::make_tuple(newHyps, logsoftmax(logits));
    }

    std::tuple<std::vector<Expr>, Expr>
    step(std::vector<Expr> hyps,
         Expr srcContext,
         Expr srcMask,
         const Beam& beam) {

      std::vector<size_t> hypIndeces;
      std::vector<size_t> embIndeces;
      std::vector<float> beamCosts;

      for(auto hyp : beam) {
        hypIndeces.push_back(hyp->GetPrevStateIndex());
        embIndeces.push_back(hyp->GetWord());
        beamCosts.push_back(hyp->GetCost());
      }

      auto graph = hyps[0]->graph();
      auto costs = graph->constant(keywords::shape={1, 1, 1, (int)beamCosts.size()},
                                   keywords::init=inits::from_vector(beamCosts));

      std::vector<Expr> newHyps;
      Expr probs;
      std::tie(newHyps, probs) = step(hyps,
                                      srcContext,
                                      srcMask,
                                      hypIndeces,
                                      embIndeces);
      probs = probs + costs;
      return std::make_tuple(newHyps, probs);
    }

    Ptr<History> search(Ptr<ExpressionGraph> graph,
                        Ptr<data::CorpusBatch> batch) {

      std::vector<Expr> startStates;
      Expr srcContext, srcMask;
      std::tie(startStates, srcContext, srcMask)
        = builder_->buildEncoder(graph, batch);

      size_t pos = 0;
      auto history = New<History>(0);
      Beam beam(1, New<Hypothesis>());
      bool first = true;
      bool final = false;
      std::vector<size_t> beamSizes(1, beamSize_);
      auto nth = New<NthElement>(beamSize_, batch->size(), stream_);

      history->Add(beam);

      std::vector<Expr> hyps;
      Expr probs;
      do {

        if(first) {
          std::tie(hyps, probs) = step(startStates,
                                       srcContext,
                                       srcMask);
          pos = graph->forward();
        }
        else {
          std::tie(hyps, probs) = step(hyps,
                                       srcContext,
                                       srcMask,
                                       beam);
          beamSizes[0] = beam.size();
          pos = graph->forward(pos);
        }

        size_t dimTrgVoc = probs->shape()[1];

        std::vector<unsigned> outKeys;
        std::vector<float> outCosts;

        for(int i = 0; i < probs->shape()[3]; i++) {
          probs->val()->set(i * dimTrgVoc + 1, std::numeric_limits<float>::lowest());
        }

        nth->getNBestList(beamSizes, probs->val(),
                          outCosts, outKeys, first);
        first = false;

        beam = toHyps(outKeys, outCosts, dimTrgVoc, beam);
        final = history->size() >= 3 * batch->words();
        history->Add(beam, final);
        beam = pruneBeam(beam);

      } while(!beam.empty() && !final);

      return history;
    }
};

}

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;

  auto options = New<Config>(argc, argv, false);

  std::vector<std::string> files =
    {"../benchmark/marian32K/newstest2016.tok.true.bpe.en"};
    //{"../benchmark/marian32K/test.txt"};

  std::vector<std::string> vocab =
    {"../benchmark/marian32K/train.tok.true.bpe.en.json"};

  YAML::Node& c = options->get();
  c["train-sets"] = files;
  c["vocabs"] = vocab;

  auto corpus = DataSet<Corpus>(options);
  BatchGenerator<Corpus> bg(corpus, options);

  auto graph = New<ExpressionGraph>();
  graph->setDevice(0);

  auto target = New<Vocab>();
  target->load("../benchmark/marian32K/train.tok.true.bpe.de.json", 50000);

  auto encdec = New<GNMT>(options, true);
  encdec->load(graph, "../benchmark/marian32K/model15.120000.npz");

  graph->reserveWorkspaceMB(128);

  boost::timer::cpu_timer timer;
  bg.prepare(false);
  while(bg) {
    auto batch = bg.next();
    auto search = New<BeamSearch<GNMT>>(encdec);
    auto history = search->search(graph, batch);

    auto results = history->NBest(1);
    for(auto r : results) {
        for(auto w : r.first)
        if(w != 0)
          std::cout << (*target)[w] << " ";
      //std::cout << r.second->GetCost() << std::endl;
      std::cout << std::endl;
    }
  }
  std::cerr << timer.format(5, "%ws") << std::endl;

  return 0;

}
