#include <iostream>
#include <cstdio>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

#define CUDNN_CALL(x) do { if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n",hipdnnGetErrorString(x),__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

int main() {
  std::cerr << "Creating CUDNN handle...";
  hipdnnHandle_t cudnnHandle;
  hipdnnCreate(&cudnnHandle);
  std::cerr << " DONE\n";

  int numBatches = 2;
  int numLayers = 3;
  int numWords = 4;
  int dimWord = 1;

  std::cerr << "Allocating sample data (embeddings)...";
  std::vector<float> embeddings(numBatches * numLayers * numWords * dimWord);
  for(int i = 0; i < numBatches * numLayers * numWords * dimWord; ++i) {
    embeddings[i] = i;
  }

  float* h_emb;
  CUDA_CALL( hipMalloc(&h_emb, sizeof(float) * numBatches *numLayers * numWords * dimWord) );
  CUDA_CALL( hipMemcpy(h_emb, embeddings.data(), sizeof(float) * numBatches * numLayers * numWords * dimWord, hipMemcpyHostToDevice) );

  hipdnnTensorDescriptor_t embTensor;
  CUDNN_CALL( hipdnnCreateTensorDescriptor(&embTensor) );
  CUDNN_CALL( hipdnnSetTensor4dDescriptor(embTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                             numBatches, numLayers, numWords, dimWord) );
  std::cerr << "input dims: " << numBatches <<  " " << numLayers << " " << numWords << " " << dimWord << std::endl;
  std::cerr << " DONE\n";

  std::cerr << "Creating and setting Conv Descriptor...";
  hipdnnConvolutionDescriptor_t convDesc;
  CUDNN_CALL( hipdnnCreateConvolutionDescriptor(&convDesc) );
  CUDNN_CALL( hipdnnSetConvolution2dDescriptor(convDesc,
      1, 0,  // padding
      1, 1,  // strides
      1, 1,  // upscales
      HIPDNN_CONVOLUTION) );

  std::cerr << " DONE\n";


  std::cerr << "Creating and setting Filters...";
  hipdnnFilterDescriptor_t filterDesc;
  CUDNN_CALL( hipdnnCreateFilterDescriptor(&filterDesc) );

  int numFilters = 1;
  int filterHeight = 1;
  int filterWidth = 1;
  int filterC = 3;

  CUDNN_CALL( hipdnnSetFilter4dDescriptor(
        filterDesc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        numFilters, filterC,
        filterHeight, filterWidth));

  std::cerr << "filter: " << numFilters << " "<< filterC << " " << filterWidth << " " << filterHeight << std::endl;

  std::vector<float> filters(numFilters * filterC * filterWidth * filterHeight);
  for (size_t i = 0; i < filters.size(); ++i) {
    filters[i] = 1;
  }
  float* d_filters;
  CUDA_CALL( hipMalloc(&d_filters, numFilters * filterC * dimWord * filterWidth * sizeof(float)) );
  CUDA_CALL( hipMemcpy(d_filters, filters.data(), filters.size() * sizeof(float), hipMemcpyHostToDevice ) );

  std::cerr << " DONE\n";

  std::cerr << "Allocating output memory...";

  int outputN, outputC, outputH, outputW;
  CUDNN_CALL( hipdnnGetConvolution2dForwardOutputDim(convDesc, embTensor, filterDesc,
              &outputN, &outputC, &outputH, &outputW) );
  std::vector<float> output(outputN * outputC * outputH * outputW);
  float* d_output;
  CUDA_CALL( hipMalloc(&d_output, outputN * outputC * outputH * outputW * sizeof(float)) );
  hipdnnTensorDescriptor_t outputDesc;
  CUDNN_CALL( hipdnnCreateTensorDescriptor(&outputDesc) );
  CUDNN_CALL( hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                             outputN, outputC, outputH, outputW) );
  std::cerr << " DONE\n";
  std::cerr << "output dim: " << outputN << " " << outputC << " " << outputH << " " << outputW << std::endl;

  float alpha = 1.0f;
  float beta = 0.0f;

  std::cerr << "Start finding conv algorithm...";

  int maxNumAlgos = 3;
  hipdnnConvolutionFwdAlgoPerf_t algos[maxNumAlgos];
  int numAlgos = -1;
  CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithm(cudnnHandle, embTensor, filterDesc, convDesc, outputDesc,
      maxNumAlgos, &numAlgos, algos) );
  std::cerr << " DONE\n";

  std::cerr << "returned algo results: " << numAlgos << std::endl;

  for (int i = 0; i < numAlgos; ++i) {
    std::cerr
      << "Algorithm: " << int(algos[i].algo) << "\t"
      << "Status: " << hipdnnGetErrorString(algos[i].status) << "\t"
      << "memory: " << algos[i].memory << "\t"
      << std::endl;
  }

  hipdnnConvolutionFwdAlgoPerf_t& bestAlgo = algos[0];


  float* d_workspace;
  int workspaceSize = bestAlgo.memory;
  CUDA_CALL( hipMalloc(&d_workspace, workspaceSize) );

  CUDA_CALL( hipSetDevice(0) );

  std::cerr << "Start forward...";

  CUDNN_CALL( hipdnnConvolutionForward(
                                cudnnHandle,
                                &alpha,
                                embTensor,
                                h_emb,
                                filterDesc,
                                d_filters,
                                convDesc,
                                bestAlgo.algo,
                                d_workspace,
                                workspaceSize,
                                &beta,
                                outputDesc,
                                d_output
  ) );

  std::cerr << " DONE\n";

  std::cerr << "Printing results: " << std::endl;

  CUDA_CALL( hipMemcpy(output.data(), d_output, sizeof(float) * outputN * outputC * outputH * outputW,
        hipMemcpyDeviceToHost) );

    for (size_t j = 0; j < output.size(); ++j) {
      std::cerr << output[j] << " ";
    }
    std::cerr << std::endl;

  return 0;
}
