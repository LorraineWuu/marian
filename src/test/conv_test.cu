#include <iostream>
#include <cstdio>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

#define CUDNN_CALL(x) do { if((x) != HIPDNN_STATUS_SUCCESS) { \
      printf("Error (%s) at %s:%d\n",hipdnnGetErrorString(x),__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

int main() {
  std::cerr << "Creating CUDNN handle...";
  hipdnnHandle_t cudnnHandle;
  hipdnnCreate(&cudnnHandle);
  std::cerr << " DONE\n";

  int numWords = 20;
  int dimWord = 100;
  int numBatches = 1;
  int numLayers = 1;

  std::cerr << "Allocating sample data (embeddings)...";
  std::vector<float> embeddings(numWords * dimWord);
  for(int i = 0; i < numWords * dimWord; ++i) {
    embeddings[i] = i;
  }

  float* h_emb;
  CUDA_CALL( hipMalloc(&h_emb, sizeof(float) * numWords * dimWord) );
  CUDA_CALL( hipMemcpy(h_emb, embeddings.data(), sizeof(float) * numWords * dimWord, hipMemcpyHostToDevice) );

  hipdnnTensorDescriptor_t embTensor;
  CUDNN_CALL( hipdnnCreateTensorDescriptor(&embTensor) );
  CUDNN_CALL( hipdnnSetTensor4dDescriptor(embTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                             numBatches, numLayers, dimWord, numWords) );
  std::cerr << " DONE\n";

  std::cerr << "Creating and setting Conv Descriptor...";
  hipdnnConvolutionDescriptor_t convDesc;
  CUDNN_CALL( hipdnnCreateConvolutionDescriptor(&convDesc) );
  CUDNN_CALL( hipdnnSetConvolution2dDescriptor(convDesc,
      0, 1,  // padding
      1, 1,  // strides
      1, 1,  // upscales
      HIPDNN_CONVOLUTION) );

  std::cerr << " DONE\n";



  std::cerr << "Creating and setting Filters...";
  hipdnnFilterDescriptor_t filterDesc;
  CUDNN_CALL( hipdnnCreateFilterDescriptor(&filterDesc) );

  int numFilters = 2;
  int filterWidth = 3;

  CUDNN_CALL( hipdnnSetFilter4dDescriptor(
        filterDesc,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        numFilters, 1,
        dimWord, filterWidth
  ) );
  std::vector<float> filters(numFilters * dimWord * filterWidth);
  for (auto& v: filters) v = 1.0f;
  float* d_filters;
  CUDA_CALL( hipMalloc(&d_filters, numFilters * dimWord * filterWidth * sizeof(float)) );
  CUDA_CALL( hipMemcpy(d_filters, filters.data(), filters.size() * sizeof(float), hipMemcpyHostToDevice ) );

  std::cerr << " DONE\n";

  std::cerr << "Allocating output memory...";

  int outputN, outputC, outputH, outputW;
  CUDNN_CALL( hipdnnGetConvolution2dForwardOutputDim(convDesc, embTensor, filterDesc,
              &outputN, &outputC, &outputH, &outputW) );
  std::vector<float> output(outputN * outputC * outputH * outputW);
  float* d_output;
  CUDA_CALL( hipMalloc(&d_output, outputN * outputC * outputH * outputW * sizeof(float)) );
  hipdnnTensorDescriptor_t outputDesc;
  CUDNN_CALL( hipdnnCreateTensorDescriptor(&outputDesc) );
  CUDNN_CALL( hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                             outputN, outputC, outputH, outputW) );
  std::cerr << " DONE\n";
  std::cerr << "output dim: " << outputN << " " << outputC << " " << outputH << " " << outputW << std::endl;

  float alpha = 1.0f;
  float beta = 0.0f;

  std::cerr << "Start finding conv algorithm...";

  int maxNumAlgos = 3;
  hipdnnConvolutionFwdAlgoPerf_t algos[maxNumAlgos];
  int numAlgos = -1;
  CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithm(cudnnHandle, embTensor, filterDesc, convDesc, outputDesc,
      maxNumAlgos, &numAlgos, algos) );
  std::cerr << " DONE\n";

  std::cerr << "returned algo results: " << numAlgos << std::endl;

  for (int i = 0; i < numAlgos; ++i) {
    std::cerr
      << "Algorithm: " << int(algos[i].algo) << "\t"
      << "Status: " << hipdnnGetErrorString(algos[i].status) << "\t"
      << "memory: " << algos[i].memory << "\t"
      << std::endl;
  }

  hipdnnConvolutionFwdAlgoPerf_t& bestAlgo = algos[0];


  float* d_workspace;
  int workspaceSize = bestAlgo.memory;
  CUDA_CALL( hipMalloc(&d_workspace, workspaceSize) );

  CUDA_CALL( hipSetDevice(0) );

  std::cerr << "Start forward...";

  CUDNN_CALL( hipdnnConvolutionForward(
                                cudnnHandle,
                                &alpha,
                                embTensor,
                                h_emb,
                                filterDesc,
                                d_filters,
                                convDesc,
                                bestAlgo.algo,
                                d_workspace,
                                workspaceSize,
                                &beta,
                                outputDesc,
                                d_output
  ) );

  std::cerr << " DONE\n";

  std::cerr << "Printing results: " << std::endl;

  CUDA_CALL( hipMemcpy(output.data(), d_output, sizeof(float) * outputN * outputC * outputH * outputW,
        hipMemcpyDeviceToHost) );

  for (size_t i = 0; i < numFilters; ++i) {
    for (size_t j = 0; j < outputH * outputW; ++j) {
      std::cerr << output[i * outputH * outputW + j] << " ";
    }
    std::cerr << std::endl;
  }

  return 0;
}
