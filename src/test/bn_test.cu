#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <functional>

#include "layers/generic.h"
#include "marian.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;
  using namespace keywords;

  auto options = New<Config>(argc, argv, false);

  int batchSize = 2;

  std::vector<float> temp(batchSize * 9);
  for (size_t i = 0; i < temp.size(); ++i) {
    temp[i] = i + 1;
  }

  std::cerr << "Building graph" << std::endl;
  {
    auto graph = New<ExpressionGraph>();
    graph->setDevice(0);
    graph->reserveWorkspaceMB(128);

    auto x = graph->param("x", {batchSize, 1, 3, 3}, init=inits::from_vector(temp));

    auto filter = graph->param("filter", {6, 1, 3, 2}, init=inits::from_value(1.0f));

    auto y = convolution(x, filter);
    auto pool = max_pooling(y);

    auto cost = sum(pool, keywords::axis=1);

    debug(y, "y");
    debug(x, "x");
    debug(pool, "pool");
    debug(cost, "cost");
    debug(filter, "filter");

    std::cerr << "Forward" << std::endl;
    graph->forward();
    std::cerr << "Backward" << std::endl;
    graph->backward();

    std::cerr << "Forward" << std::endl;
    graph->forward();
    std::cerr << "Backward" << std::endl;
    graph->backward();

    std::vector<float> tmp(50);
    filter->grad() >> tmp;

  }

  return 0;
}
