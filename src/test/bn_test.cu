#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <functional>

#include "layers/generic.h"
#include "marian.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;
  using namespace keywords;

  auto options = New<Config>(argc, argv, false);

  int batchSize = 1;

  std::vector<float> temp(batchSize * 9);
  for (size_t i = 0; i < temp.size(); ++i) {
    temp[i] = i + 1;
  }
  std::vector<float> temp2(3072 * 3072);
  std::vector<float> indeces(batchSize, 0.f);


  std::cerr << "Building graph" << std::endl;
  {
    auto graph = New<ExpressionGraph>();
    graph->setDevice(0);
    graph->reserveWorkspaceMB(128);

    std::cerr << "Setting X" << std::endl;
    auto x = graph->param("x", {batchSize, 1, 3, 3}, init=inits::from_vector(temp));

    std::cerr << "Setting filter" << std::endl;
    auto filter = graph->param("gamma", {16, 1, 3, 2}, init=inits::from_value(1.0f));

    std::cerr << "Setting convolution" << std::endl;
    auto y = convolution(x, filter);
    auto pool = max_pooling(y);

    auto idx = graph->constant(shape={16, 1},
                               init=inits::from_value(1.0f));
    auto ce = cross_entropy(pool, idx);
    auto cost = mean(sum(ce, keywords::axis=1), keywords::axis=1);

    debug(y, "y");
    debug(x, "x");
    debug(ce, "ce");
    debug(pool, "pool");
    debug(cost, "cost");
    debug(filter, "filter");

    std::cerr << "Forward" << std::endl;
    graph->forward();
    std::cerr << "Backward" << std::endl;
    graph->backward();
  }

  return 0;
}
