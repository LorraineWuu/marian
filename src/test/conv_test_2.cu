#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <functional>

#include "layers/convolution.h"
#include "marian.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;
  using namespace keywords;

  auto options = New<Config>(argc, argv, false);

  int numWords = 4;
  int dimWord = 3;
  int numBatches = 2;
  int numLayers = 1;

  std::vector<float> embData(numBatches * numWords * dimWord);
  std::vector<float> embMask(numBatches * numWords * dimWord);

  for (size_t i = 0; i < embData.size(); ++i) {
    embData[i] = i;
    embMask[i] = 1;
  }

  std::cerr << "Building graph" << std::endl;
  {
    auto graph = New<ExpressionGraph>();
    graph->setDevice(0);
    graph->reserveWorkspaceMB(128);

    auto x = graph->param("x", {numBatches, dimWord, numWords}, init=inits::from_vector(embData));

    auto xMask = graph->constant(shape={numBatches, 1, numWords}, init=inits::from_vector(embMask));

    auto convLayer = MultiConvLayer("enc_", 2) (x, xMask);

    auto cost = sum(convLayer, keywords::axis=1);

    debug(x, "x");
    debug(convLayer, "conv");
    /* debug(cost, "cost"); */

    std::cerr << "Forward" << std::endl;
    graph->forward();
    hipDeviceSynchronize();
    std::vector<float> tmp;
    convLayer->val() >> tmp;
    for (auto f :  tmp) std::cerr << f  << " ";
    std::cerr << std::endl;
    std::cerr << "Backward" << std::endl;
    graph->backward();
    tmp.clear();
    convLayer->grad() >> tmp;
    hipDeviceSynchronize();
    std::cerr << "kon\n";
    for (auto f :  tmp) std::cerr << f  << " ";
    std::cerr << std::endl;
  }

  return 0;
}
