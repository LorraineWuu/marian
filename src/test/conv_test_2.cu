#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <cmath>
#include <random>
#include <algorithm>
#include <iterator>
#include <iostream>
#include <functional>

#include "layers/convolution.h"
#include "marian.h"

int main(int argc, char** argv) {
  using namespace marian;
  using namespace data;
  using namespace keywords;

  auto options = New<Config>(argc, argv, false);

  int batchSize = 2;

  std::vector<float> temp(batchSize * 9);
  for (size_t i = 0; i < temp.size(); ++i) {
    temp[i] = i + 1;
  }

  const int numKernels = 6;
  const int kernelHeight = 3;
  const int kernelWidth = 2;

  std::cerr << "Building graph" << std::endl;
  {
    auto graph = New<ExpressionGraph>();
    graph->setDevice(0);
    graph->reserveWorkspaceMB(128);

    auto x = graph->param("x", {batchSize, 1, 3, 3}, init=inits::from_vector(temp));

    auto convLayer = Convolution("conv_layer")(x, numKernels, kernelHeight, kernelWidth, 0,0,
        -1, -1, 0, 0);

    /* auto y = convolution(x, filter); */
    /* auto pool = max_pooling(y); */

    auto cost = sum(convLayer, keywords::axis=1);

    debug(x, "x");
    debug(convLayer, "conv");
    debug(cost, "cost");

    std::cerr << "Forward" << std::endl;
    graph->forward();
    std::cerr << "Backward" << std::endl;
    graph->backward();
  }

  return 0;
}
